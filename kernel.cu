#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include "renderer.h"
#include <stdlib.h>
#include <chrono>
#include <iostream>
#include <random>

//opengl viewport boundaries
#define min_x -1.0f
#define max_x 1.0f
#define min_y -1.0f
#define max_y 1.0f

#define PI 3.1415926535f

#define numberOfParticles 1000
#define particleRadius 0.001f
#define particleMass 1.0f

#define damping 0.3f
#define gravity 9.8f
#define gravityModifier 0.1f


__host__ void particles_init(ParticleCenter* h_pos, ParticleCenter* h_vel) {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<float> dis(-1.0f, 1.0f);

  for (int i = 0; i < numberOfParticles; i++) {
	  h_pos[i].x = dis(gen);
    h_pos[i].y = dis(gen);
    h_vel[i].x = dis(gen);
    h_vel[i].y = dis(gen);
  }
}

__device__ float smoothingKernel(float radius, float distance) {
  if(distance > radius) return 0.0f;
  float value = (radius - distance) * (radius - distance) * (radius - distance);
  float volume = PI * std::pow(radius - distance, 5) / 10;
  return value / volume;
}

__device__ float calcDistance(ParticleCenter p1, ParticleCenter p2) {
  return sqrt((p1.x - p2.x) * (p1.x - p2.x) + (p1.y - p2.y) * (p1.y - p2.y));
}

__device__ float calcDensity(int id, ParticleCenter* h_pos){
  float density = 0.0f;

  for(int i = 0; i < numberOfParticles; i++){
    if (i == id) continue;
    float distance = calcDistance(h_pos[id], h_pos[i]);
    float influence = smoothingKernel(particleRadius, distance);
    density += particleMass * smoothingKernel(particleRadius, distance);
  }
  return density;
}

__global__ void update(ParticleCenter* pos, ParticleCenter* vel, float deltaMicro) {
  float deltaSeconds = deltaMicro / 1000000;
  int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < numberOfParticles) {
    //collision with the floor/ceiling
    if(pos[i].y <= min_y + particleRadius) {
      pos[i].y = min_y + particleRadius;
      vel[i].y = damping * -vel[i].y;
    }else if(pos[i].x >= max_y - particleRadius) {
      pos[i].y = max_y - particleRadius;
      pos[i].y = damping * -vel[i].y;
    }

    //collision with the walls
    if(pos[i].x <= min_x + particleRadius) {
      pos[i].x = min_x + particleRadius;
      vel[i].x = damping * -vel[i].x;
    }else if(pos[i].x >= max_x - particleRadius) {
      pos[i].x = max_x - particleRadius;
      vel[i].x = damping * -vel[i].x;
    }

    //updating the velocity
    vel[i].y += ((-gravity * gravityModifier)/particleMass) * deltaSeconds;

    // updating the position
    pos[i].x = pos[i].x + vel[i].x * deltaSeconds;
    pos[i].y = pos[i].y + vel[i].y * deltaSeconds;
	}
}

int main() {
  const unsigned int WIDTH = 1920;
  const unsigned int HEIGHT = 1080;

	//simulation parameters
	int NUM_THREADS = 256;
	int NUM_BLOCKS = (numberOfParticles + NUM_THREADS - 1) / NUM_THREADS;

  // int gridX = std::ceil(2.0f / particleRadius);
  // int gridY = std::ceil(2.0f / particleRadius);

	//host vector pointers
	ParticleCenter* h_pos, * h_vel;
  ParticleCenter* h_grid;

	//device vector pointers
	ParticleCenter* d_pos, * d_vel;
  ParticleCenter* * d_grid;
	//size of the vectors in bytes
	size_t bytes = numberOfParticles * sizeof(ParticleCenter);
  // size_t gridBytes = gridX * gridY * sizeof(float);

	//allocate memory for the host vectors
	h_pos = (ParticleCenter*)malloc(bytes);
	h_vel = (ParticleCenter*)malloc(bytes);
  // h_grid = (ParticleCenter*)malloc(gridBytes);

	//allocate memory for the device vectors
	hipMalloc(&d_pos, bytes);
	hipMalloc(&d_vel, bytes);

	//initialize the particles
	particles_init(h_pos, h_vel);

	//initialize opengl renderer
	Renderer renderer;
  
  //triangle length from inner circle radius
  float triangleLength = particleRadius * 2 * sqrt(3);
  renderer.init(WIDTH, HEIGHT, triangleLength);

  std::chrono::steady_clock::time_point timer = std::chrono::steady_clock::now();
  std::chrono::duration<float, std::micro> microDif;
	while (renderer.render(h_pos, 2 * numberOfParticles)) {
		//copy the host vectors to the device vectors
		hipMemcpy(d_pos, h_pos, bytes, hipMemcpyHostToDevice);
		hipMemcpy(d_vel, h_vel, bytes, hipMemcpyHostToDevice);

		//update the particles
    microDif = std::chrono::steady_clock::now() - timer;
    timer = std::chrono::steady_clock::now();
		update<<<NUM_BLOCKS, NUM_THREADS>>> (d_pos, d_vel, microDif.count());

		//copy the device vectors to the host vectors
		hipMemcpy(h_pos, d_pos, bytes, hipMemcpyDeviceToHost);
		hipMemcpy(h_vel, d_vel, bytes, hipMemcpyDeviceToHost);
	}

	//free memory
	free(h_pos);
	free(h_vel);
	hipFree(d_pos);
	hipFree(d_vel);

	return 0;
}